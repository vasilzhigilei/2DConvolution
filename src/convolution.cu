#include "hip/hip_runtime.h"
//============================================================================
// 2D Convolution
// A CUDA 2D convolution implementation for GPGPUs
//
// Main file for testing and running the CUDA kernel
//
// Written by:    Vasil Zhigilei
//============================================================================

// includes, system
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

using namespace std;

// texture input memory
texture<int, 2, hipReadModeElementType> texIn;

// constant filter memory
__constant__ int filter[9];

// number of threads per block
const int THREADS_PER_BLOCK = 256;

__global__ void convolution1(int *output, int DIM){
	/*
	 * 2D convolution using constant filter memory and texture input memory optimizations
	 */
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	// check if thread within DIM bounds
	if(x < DIM && y < DIM){
		// initialize dot product
		int dot_product = 0;
		for(int i = 0; i < 3; i++){
			for(int j = 0; j < 3; j++){
				// compute dot product with input data and filter
				dot_product += tex2D(texIn, x + i - 1, y + j - 1) * filter[i + j*3];
			}
		}
		// copy dot product to device output memory
		output[offset] = dot_product;
	}
}

int main(){
	/*
	 * Main function for running the 2D convolution kernel
	 */

	// dimension of one side of the 2D input data
	int DIM = 1024;
	// memory size required for input
	int SIZE = DIM * DIM * sizeof(int);

	// seed generation for random, use same seed for same results every run
	srand(10);

	// host memory allocation
	int *h_input = (int *) malloc(SIZE);
	int *h_output = (int *) malloc(SIZE);
	int *h_filter = (int *) malloc(9*sizeof(int));

	// device memory allocation
	int *d_input, *d_output;
	hipMalloc((void**) &d_input, SIZE);
	hipMalloc((void**) &d_output, SIZE);

	// texture memory binding to input
	hipChannelFormatDesc desc = hipCreateChannelDesc<int>();
	hipBindTexture2D(0, &texIn, d_input, &desc, DIM, DIM, DIM*sizeof(int));

	// generate input data
	cout << "Generating input data..." << endl;
	for(int i = 0; i<DIM; i++){
		for(int j = 0; j<DIM; j++){
			h_input[i+j*DIM] = rand() % 11 - 5;
		}
	}

	// copy input memory to device
	hipMemcpy(d_input, h_input, SIZE, hipMemcpyHostToDevice);

	// Generate filter
	cout << "Generating filter..." << endl;
	for(int i = 0; i<3; i++){
		for(int j = 0; j<3; j++){
			h_filter[i+j*3] = rand() % 11 - 5;
		}
	}

	// copy filter to constant device memory
	hipMemcpyToSymbol(HIP_SYMBOL(filter), h_filter, 9*sizeof(int));

	// using THREADS_PER_BLOCK, calculate grid and block size
	dim3 grid_size(16*((DIM+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK), 16*((DIM+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK), 1);
	dim3 block_size(THREADS_PER_BLOCK/16, THREADS_PER_BLOCK/16, 1);

	// run convolution kernel
	convolution1<<<grid_size, block_size>>>(d_output, DIM);

	// copy results back to host
	hipMemcpy(h_output, d_output, SIZE, hipMemcpyDeviceToHost);

	// console output some of the outputs
	for(int i = 0; i<DIM*DIM-1; i+=DIM*DIM/10){
		cout << i << ": " << h_output[i] << endl;
	}



	// clean up memory on host & device
	free(h_input);
	free(h_output);
	free(h_filter);

	hipUnbindTexture(texIn);

	hipFree(d_input);
	hipFree(d_output);

}
